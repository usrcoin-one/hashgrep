#include "hip/hip_runtime.h"
/* -*- Mode: C++; c-basic-offset: 4; indent-tabs-mode: nil -*- */

/**
   Copyright 2012 Carnegie Mellon University

   Authors: Bin Fan, Iulian Moraru and David G. Andersen

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

   This is the implementation of a hash based grep for GPGPU.
*/

#define _DARWIN_FEATURE_64_BIT_INODE 1
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/uio.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/mman.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math_functions.h"

#include "sbox.h"
#include "cuckoohashtable.h"

using namespace std;


extern "C" {
    int getfile(char *infile, size_t *filesize);
#include "timing.h"
}

#define BLOCK_SIZE 256
#define HASH_LEN 19
#define FILE_MAX 6710886400
#define NR_STREAMS 10
//#define NR_STREAMS 1

#ifndef DEF_STRBUF_SIZE
#define DEF_STRBUF_SIZE 0xa000000
#endif

//#define COUNTING
#define FILTERING

char *pinnedBuf;
char *strbuf;
size_t strbuf_used = 1;


#define num_indexbits 20
#define num_tagbits 16
#define num_bfbits 128

typedef CuckooHashtable<uint64_t, uint32_t, num_tagbits, num_indexbits, num_bfbits> HashfilterType;
typedef HashfilterType::Bucket BucketType;

const uint32_t bucket_size = 4;
const uint32_t num_buckets = 1ULL << num_indexbits;
const uint32_t INDEXMASK = num_buckets - 1;
const uint32_t TAGMASK = (1ULL << num_tagbits) - 1;
const uint32_t BFINDEXMASK  = num_bfbits - 1;

HashfilterType hashfilter;

unsigned char *dev_buckets;
unsigned char *dev_strbuf;
unsigned char *dev_corpus;
unsigned char *dev_pos_bitmap;
unsigned int *dev_cnt1;
unsigned int *dev_cnt2;

void exit_on_error(const char *name, hipError_t err) {
    if (err) {
        if (err) printf("%s Error: %s\n", name, hipGetErrorString(err));
        exit(-1);
    }
}

inline uint32_t  rol32(unsigned int word, int shift)
{
    return (word << shift) | (word >> (32 - shift));
}

inline __device__ uint32_t  dev_rol32(unsigned int word, int shift)
{
    return (word << shift) | (word >> (32 - shift));
}

void setup(char* pattern_file) 
{
    ifstream inPhrases(pattern_file);
    if (!inPhrases.is_open()) {
        cerr << "Can not open phrase file " << pattern_file << endl;
        exit(-1);
    }

    string pattern;
    while (getline(inPhrases, pattern)) {
        if (pattern.length() < HASH_LEN) {
            perror("Search phrase too short");
            exit(-1);
        }

        uint32_t  hval[2] = {0, 0};

        for (int j = 0; j < HASH_LEN; j++) {
            hval[0] = rol32(hval[0], 1);
            hval[1] = rol32(hval[1], 3);
            uint32_t  sbv = host_sbox[pattern[j]];
            hval[0] ^= sbv;
            hval[1] ^= sbv;
        }
        
        // printf("\n");
        // printf("hal={%x %x}\n", hval[0], hval[1]);

        uint32_t p;
        if (hashfilter.Get(*((uint64_t*) hval), p) == Ok) {
            assert(p > 0);
        } else
            p = 0;

        if (strbuf_used >= DEF_STRBUF_SIZE)  {
            perror("Not enough strbuf, please make DEF_STRBUF_SIZE larger ");
        }

        char* cstr = strbuf + strbuf_used;
        memcpy(cstr, &p, sizeof(uint32_t));
        strcpy(cstr + sizeof(uint32_t), pattern.c_str());

        if (hashfilter.Put(*((uint64_t*) hval), strbuf_used) != Ok) {
            perror("Error while buiding hash table");
            exit(-1);
        }
        strbuf_used += pattern.size() + sizeof(uint32_t) + 1;
            
    }
    inPhrases.close();
   
    hashfilter.BuildBF();

    size_t len = hashfilter.SizeInBytes();

    exit_on_error("hipMalloc",
                  hipMalloc((void **) &dev_buckets, 
                             len));

    exit_on_error("hipMalloc",
                  hipMalloc((void **) &dev_strbuf, 
                             strbuf_used));

    exit_on_error("hipMemcpy",
                  hipMemcpy(dev_buckets, 
                             hashfilter.buckets_, 
                             len, 
                             hipMemcpyHostToDevice));

    exit_on_error("hipMemcpy",
                  hipMemcpy(dev_strbuf, 
                             strbuf, 
                             strbuf_used, 
                             hipMemcpyHostToDevice));


#ifdef COUNTING
    exit_on_error("hipMalloc dev_cnt1",
                  hipMalloc((void **) &dev_cnt1, 
                             sizeof(unsigned int)));


    exit_on_error("hipMalloc dev_cnt2",
                  hipMalloc((void **) &dev_cnt2, 
                             sizeof(unsigned int)));

    exit_on_error("hipMemset dev_cnt1 = 0",
                  hipMemset(dev_cnt1, 0, sizeof(unsigned int)));

    exit_on_error("hipMemset dev_cnt2 = 0",
                  hipMemset(dev_cnt2, 0, sizeof(unsigned int)));

#endif

}

void test_setup(char* pattern_file) 
{

    cerr<< "testing setup result" << endl;
    ifstream inPhrases(pattern_file);
    if (!inPhrases.is_open()) {
        cerr << "Can not open phrase file " << pattern_file << endl;
        exit(-1);
    }

    string pattern;
    while (getline(inPhrases, pattern)) {
        if (pattern.length() < HASH_LEN) {
            perror("Search phrase too short");
            exit(-1);
        }

        uint32_t  hval[2] = {0, 0};

        for (int j = 0; j < HASH_LEN; j++) {
            hval[0] = rol32(hval[0], 1);
            hval[1] = rol32(hval[1], 3);
            uint32_t  sbv = host_sbox[pattern[j]];
            hval[0] ^= sbv;
            hval[1] ^= sbv;
        }
            
        uint32_t p;
        if (hashfilter.Get(*((uint64_t*) hval), p) == Ok) {
            continue;
        } 

        cout << "cannot find " << pattern;
            
    }
    inPhrases.close();
   
}

void cleanup() 
{
    hipFree(dev_strbuf);
    hipFree(dev_buckets);
    hipHostFree(pinnedBuf);

#ifdef COUNTING
    hipFree(dev_cnt1);
    hipFree(dev_cnt2);
#endif

}

int pick_dim(dim3 &dimGrid,
             dim3 &dimBlock,
             int numthreads,
             int blocksize)
{
    unsigned int blocks_y = 1;
    unsigned int blocks_x = 1;
    unsigned int threads_1d = numthreads % blocksize;

    if (numthreads > (256 * blocksize)) {
        blocks_x = 256;
        blocks_y = numthreads / (256 * blocksize);
        threads_1d = blocksize;
    } else if (numthreads > blocksize) {
        blocks_x = numthreads / blocksize;
        blocks_y = 1;
        threads_1d = blocksize;
    } else {
        blocks_x = 1;
        blocks_y = 1;
        threads_1d = numthreads;
    }

    unsigned int threads_used = blocks_y * blocks_x * threads_1d;
    numthreads -= threads_used;
    printf("pick_dim %d %d %d\n", blocks_y, blocks_x, threads_1d);
    dimGrid = dim3(blocks_x, blocks_y);
    dimBlock = dim3(threads_1d);
    return threads_used;
}


inline __device__ void dev_set_bit(int i, unsigned char *bv) {
    unsigned int *p = (unsigned int *)bv;
    unsigned int bitMask = 1 << (i & 31);
    atomicOr(&p[i >> 5], bitMask);
}

inline __device__ bool dev_is_bit_set(int i, unsigned char *bv) {
    unsigned int bitMask = 1 << (i & 7);
    return (bv[i >> 3] & bitMask);
}

inline bool is_bit_set(int i, unsigned int *bv) {
    unsigned int word = bv[i >> 5];
    unsigned int bitMask = 1 << (i & 31);
    return (word & bitMask);
}

__global__ void GrepKernel(unsigned char *d_a,
                           unsigned char *dev_buckets,
                           unsigned char *dev_pos_bitmap,
                           unsigned int char_offset,
                           unsigned int* dev_cnt1,
                           unsigned int* dev_cnt2)
{
    __shared__ unsigned boxed[BLOCK_SIZE + HASH_LEN];

    int i = char_offset + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.x;

    boxed[threadIdx.x] = sbox[d_a[i]];
    

    /* Ugly, but let some threads pull in the remainder */
    /* TIME:  0.01 seconds */
    int otid = threadIdx.x;
    if (otid < HASH_LEN) {
        int new_i = blockDim.x + i;
        int new_b = blockDim.x + otid;
        boxed[new_b] = sbox[d_a[new_i]];
    }

    /* TIME:  Almost none.  */
    __syncthreads();

    uint32_t  hval[2] = {0, 0};
    /* Step 2:  Compute the hash of the next HASH_LEN characters */
    for (int j = 0; j < HASH_LEN; j++) {
        hval[0] = dev_rol32(hval[0], 1);
        hval[1] = dev_rol32(hval[1], 3);
        uint32_t sbv = boxed[threadIdx.x+j];
        hval[0] ^= sbv;
        hval[1] ^= sbv;

        // if (i == 0) {
        //     printf("%c(%d), hval={%x %x}, sbv = %x\n", d_a[i+j], d_a[i+j], hval[0], hval[1], sbv);
        // }

    }


    uint32_t i1, i2, tag;
    
    tag =  hval[0] & TAGMASK;
    tag += (tag == 0); 
    i1 = hval[1] & INDEXMASK;
    i2 = (i1 ^ (tag * 0x5bd1e995)) & INDEXMASK;
    
    // if (i == 0) {
    //     printf("idx:%d\ti1 = %05x,i2 = %05x, tag=%04x\n", i, i1, i2, tag);
    // }
    // read first 24 bytes from bucket i1
    // including 8-byte tagbits and 16-byte bfbits
    ulong3 v1 = *(ulong3 *) (dev_buckets + i1 * sizeof(BucketType));
    BucketType* b1 = (BucketType*) &v1;

    uint64_t tagbits1 = *(uint64_t*) (b1->tagbits_);
    unsigned char* bfbits1 = b1->bfbits_;

    for (int j = 0; j < bucket_size; j ++) {
        uint32_t tag1 = tagbits1 & TAGMASK;
        tagbits1 >>= num_tagbits;
        if (tag1 == tag) {
            dev_set_bit(i, dev_pos_bitmap);
            return;
        }
    }



#ifdef COUNTING
    atomicAdd(dev_cnt1, 1);
#endif

#ifdef FILTERING
    if (!(dev_is_bit_set(i2 & BFINDEXMASK, bfbits1) &&
          dev_is_bit_set((i2 / num_bfbits) & BFINDEXMASK, bfbits1))) 
//          dev_is_bit_set(tag & BFINDEXMASK, bfbits1))) 
        return;
#endif

#ifdef COUNTING
    atomicAdd(dev_cnt2, 1);
#endif

    // read first 8 bytes from bucket i1
    // including 8-byte tagbits 
    ulong1 v2 = *(ulong1 *) (dev_buckets + i2 * sizeof(BucketType));
    BucketType *b2 = (BucketType*) &v2;

    uint64_t tagbits2 = *(uint64_t*) (b2->tagbits_);

    for (int j = 0; j < bucket_size; j ++) {
        uint32_t tag2 = tagbits2 & TAGMASK;
        tagbits2 >>= num_tagbits;
        if (tag2 == tag) {            
            dev_set_bit(i, dev_pos_bitmap);
            return;
        }
    }
    return;
}

void process_corpus(char* corpus_file) 
{
    printf("opening corpus file %s\n", corpus_file);
    size_t filesize;
    int f = getfile(corpus_file, &filesize);
    if (f == -1) {
        perror(corpus_file);
        exit(-1);
    }

    filesize = min((unsigned long long)filesize, (unsigned long long)FILE_MAX);

    exit_on_error("hipHostMalloc pinnedBuf", 
                  hipHostMalloc((void **)&pinnedBuf, filesize));

    exit_on_error("hipMalloc dev_corpus",
                  hipMalloc((void **)&dev_corpus, filesize + HASH_LEN));

    exit_on_error("hipMalloc dev_pos_bitmap",
                  hipMalloc((void **)&dev_pos_bitmap, filesize / 8 + 1));

    exit_on_error("hipMemset dev_pos_bitmap = 0",
                  hipMemset(dev_pos_bitmap, 0, filesize/8 + 1));


    int numthreads;
    dim3 dimGrid, dimBlock;

    hipStream_t streams[NR_STREAMS];

    for (int i = 0; i < NR_STREAMS; i++) {
        exit_on_error("hipStreamCreate",
                      hipStreamCreate(&streams[i]));
    }

    int size = filesize / NR_STREAMS;

    int fd = open(corpus_file, O_RDONLY);

    for (int i = 0; i < NR_STREAMS; i++) {
        unsigned offset = i * size;
        if (i == NR_STREAMS - 1) {
            size = filesize - i * size;
        }
        numthreads = size;

        //printf("Executing grep on %d\n", size);

        read(fd, pinnedBuf + offset, size);

        exit_on_error("hipMemcpyAsync",
                      hipMemcpyAsync(dev_corpus + offset, pinnedBuf + offset, size, hipMemcpyHostToDevice, streams[i]));

        unsigned int char_offset = 0;
        while (numthreads > 0) {
            unsigned int tu = pick_dim(dimGrid, dimBlock, numthreads, BLOCK_SIZE);
            printf("Executing GrepKernel (%d,%d,%d) @ %u\n", dimGrid.x, dimGrid.y, dimBlock.x, (offset + char_offset));
            GrepKernel<<<dimGrid, dimBlock, 0, streams[i]>>>(dev_corpus, 
                                                             dev_buckets,
                                                             dev_pos_bitmap, 
                                                             offset + char_offset,
                                                             dev_cnt1,
                                                             dev_cnt2);


            //checkReportCudaStatus("GrepKernel");
            numthreads -= tu;
            char_offset += tu;
        }
    }

    hipDeviceSynchronize();

    for (int i = 0; i < NR_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
    close(f);
    close(fd);

#ifdef COUNTING

    unsigned int host_cnt1, host_cnt2;
    exit_on_error("hipMemcpy corpus results to host",
                  hipMemcpy(&host_cnt1, dev_cnt1,
                             sizeof(unsigned int), hipMemcpyDeviceToHost));

    exit_on_error("hipMemcpy corpus results to host",
                  hipMemcpy(&host_cnt2, dev_cnt2,
                             sizeof(unsigned int), hipMemcpyDeviceToHost));

    printf("total = %u, cnt1 = %u, cnt2 = %u\n", filesize, host_cnt1, host_cnt2);
#endif

}


void print_positions(char *corpus_file)
{
    size_t filesize;
    int f = getfile(corpus_file, &filesize);
    if (f == -1) {
        perror(corpus_file);
        exit(-1);
    }
    char *buf = pinnedBuf;

    filesize = min((unsigned long long)filesize, (unsigned long long)FILE_MAX);

    unsigned int *bv = (unsigned int *)malloc(filesize / 8 + 1);

    exit_on_error("hipMemcpy corpus results to host",
                  hipMemcpy(bv, dev_pos_bitmap,
                             filesize / 8, hipMemcpyDeviceToHost));

    int file_ints = filesize / 32;
    int prev_end = -1;
    for (int i = 0; i < file_ints; i++) {
        if (bv[i]) {
	        for (int j = ffs(bv[i]) - 1; j < 32; j++) {
	            int offset = i*32 + j;
	            if (is_bit_set(offset, bv)) {
                    //cout << 32 * i + j << " ";
            
	                /* Find end of previous line */
	                if (offset > prev_end && buf[offset] != '\n') {
                        char *sol = ((char*)memrchr(buf, '\n', offset));
                        int start_line;
                        if (sol) {
                            start_line = sol - buf;
                        } else {
                            start_line = 0;
                        }

	                    int end_line;
                        char *eol = (char*)memchr(buf + offset, '\n', filesize - offset + 1);
                        end_line = eol - buf;
                        j += end_line - offset;
	                    if (buf[start_line] == '\n') start_line++;
	                    fwrite(buf + start_line, 1, end_line - start_line, stdout);
	                    fputc('\n', stdout);
	                    prev_end = end_line;
	                }
	            }
	        }
        }
    }

    close(f);
}


int main(int argc, char **argv)
{
    if (argc != 3)
    {
        cerr << "usage: " << argv[0] << " patterns corpus" << endl;
        return -1;
    }

    char *patterns_file = argv[1];
    char *corpus_file = argv[2];


    //cout << sizeof(HashfilterType::Bucket) << endl;
    //cout << sizeof(uint4) << " " << sizeof(ulong4) << endl;
    //return 0;

    strbuf = new char[DEF_STRBUF_SIZE];

    timing_stamp("start", false);

    setup(patterns_file);
    //test_setup(patterns_file);

    timing_stamp("setup", false);

    process_corpus(corpus_file);

    timing_stamp("grep corpus", false);

    print_positions(corpus_file);

    timing_stamp("copying out", false);

    cleanup();

    timing_stamp("cleanup done", true);

    timing_report();

    struct hipDeviceProp_t cdp;
    hipGetDeviceProperties(&cdp, 0);
    printf("\ndeviceOverlap = %d\n", cdp.deviceOverlap);
}

