#include "hip/hip_runtime.h"
/* -*- Mode: C++; c-basic-offset: 4; indent-tabs-mode: nil -*- */

/**
Copyright 2012 Carnegie Mellon University

Authors: Bin Fan, Iulian Moraru and David G. Andersen

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

     http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.

This is the implementation of a hash based grep for GPGPU.
*/

#define _DARWIN_FEATURE_64_BIT_INODE 1
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <fcntl.h>
 #include <unistd.h>
#include <sys/types.h>
#include <sys/uio.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/mman.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math_functions.h"

#include "sbox.h"

using namespace std;

extern "C" {
int getfile(char *infile, size_t *filesize);
#include "timing.h"
}

//#define BLOOMBITS 1048576 /* 1 millllion bits */
#define BLOOMBITS 0x10000000 /* 32 MB */
#define BLOOMMASK (BLOOMBITS - 1)

#define BLOCK_SIZE 256
#define HASH_LEN 19
#define FILE_MAX 6710886400
#define NR_STREAMS 10

char *pinnedBuf;


struct countmap {
    unsigned int hval;
    unsigned int charloc;
};

texture<unsigned char, 1, hipReadModeElementType> tex_bloom;

bool is_bit_set(int i, unsigned int *bv) {
    unsigned int word = bv[i >> 5];
    unsigned int bitMask = 1 << (i & 31);
    return (word & bitMask);
}

__device__ bool texbf_is_bit_set(int i) {
    unsigned char word = tex1Dfetch(tex_bloom, i/8);
    unsigned int bitMask = 1 << (i % 8);
    return (word & bitMask);
}

__device__ bool device_is_bit_set(int i, unsigned int *bv) {
    unsigned int word = bv[i >> 5];
    unsigned int bitMask = 1 << (i & 31);
    return (word & bitMask);
}

__device__ void device_set_bit(int i, unsigned int *bv) {
    unsigned int bitMask = 1 << (i & 31);
    atomicOr(&bv[i >> 5], bitMask);
}

inline __device__ unsigned int rol32(unsigned int word, int shift)
{
    return (word << shift) | (word >> (32 - shift));
}

__global__ void grepSetup(unsigned char *d_a,
                          unsigned int *d_b,
                          unsigned int starting_offset)
{
    /* SPEED:  Copy into local memory coalescing and then do this
     * all locally. */
    int i = starting_offset + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.x;

    int char_offset = i * (HASH_LEN + 1); /* Skip \n */

    unsigned int hval = 0, hval2 = 0;

    for (int j = 0; j < HASH_LEN; j++) {
        hval = rol32(hval, 1);
        hval2 = rol32(hval2, 3);
        unsigned int sbv = sbox[d_a[char_offset + j]];
        hval ^= sbv;
        hval2 ^= sbv;
    }
    device_set_bit(hval & BLOOMMASK, d_b);
    device_set_bit(hval2 & BLOOMMASK, d_b);
    unsigned int hval3 = hval + hval2;
    device_set_bit(hval3 & BLOOMMASK, d_b);
    unsigned int hval4 = hval + 5 * hval2;
    device_set_bit(hval4 & BLOOMMASK, d_b);
//    unsigned int hval5 = (hval << 16) | (hval2 >> 16);
//    device_set_bit(hval5 & BLOOMMASK, d_b);
}

__global__ void GrepKernel(unsigned char *d_a,
                           unsigned int *blooms,
                           unsigned int *dev_reverse_bloom,
                           unsigned int *dev_positions_matched,
                           unsigned int char_offset,
                           unsigned int n_chars)
{
    __shared__ unsigned boxed[BLOCK_SIZE + HASH_LEN];

    int i = char_offset + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.x;

    /* SPeed:  This part takes .06 seconds.  Without boxing and
     * without cleanup, it takes .04.  Without cleanup, .05. */
    /* Step 1:  Bring the base chars in to local memory,
     * sboxing them on the way in.  SPEED:  This is faster or equiv to
     * doing 32 bit reads into a register and then shifting out the
     * chars. */
    /* TIME:  0.03 seconds */
    boxed[threadIdx.x] = sbox[d_a[i]];

    /* Ugly, but let some threads pull in the remainder */
    /* TIME:  0.01 seconds */
    int otid = threadIdx.x;
    if (otid < HASH_LEN) {
        int new_i = blockDim.x + i;
        int new_b = blockDim.x + otid;
        boxed[new_b] = sbox[d_a[new_i]];
    }

    /* TIME:  Almost none.  */
    __syncthreads();

    unsigned int hval = 0, hval2 = 0;

    /* Step 2:  Compute the hash of the next HASH_LEN characters */
    for (int j = 0; j < HASH_LEN; j++) {
        hval = rol32(hval, 1);
        hval2 = rol32(hval2, 3);
        unsigned int sbv = boxed[threadIdx.x+j];
        hval ^= sbv;
        hval2 ^= sbv;
    }


    /* Other idea:  Steal from the blocked bloom filter idea to do two
     * bit lookups in a single bus transaction. */

    /* Attempt X:  Loop over the bit vector, load into local memory,
     * do a subset of the tests.  */


    /* Idea:  Have 4 threads process each character position.
     * And have them only do the bit lookup if hash >> [all but 2 bits]== <index>
     * in some way -- thus forcing locality.  Trading bandwidth, but hey,
     * we've got bandwidth.
     * To really do this right, we might want to optimize the
     * hash computation further so that we don't use too much
     * global bandwidth copying the post-hash results out.
     * XXX - probably not too helpful;  tried using lowest of 4
     * hash functions to improve texture locality, little benefit.  Maybe
     * could combine. */


    /* SPEED: This step takes 0.22 of 0.27 seconds */
    /* searchbig:  0.31 out of 0.37 */
    /* 3 version 1:  Do them into global memory and let threads diverge... */

    /* Unrolling and doing a no-branch, dual fetch is slower. */

    /* Hm.  With more hash functions, might be able to use the sorted
     * hash trick to improve locality at the cost of a bit more
     * computation.  Can we bubble sort 5 hash functions rapidly?  Does
     * that give us a cache advantage with texture memory? */

    unsigned int h1 = hval & BLOOMMASK;
    unsigned int h2 = hval2 & BLOOMMASK;
    unsigned int h3 = (hval + hval2) & BLOOMMASK;
    unsigned int h4 = (hval + 5 * hval2) & BLOOMMASK;
//    unsigned int h5 = ((hval << 16) | (hval2 >> 16)) & BLOOMMASK;

    /* This doesn't help with two hash functions */
    /* Kernel time:  0.38 with, 0.37 without */

    unsigned int w1 = h1 >> 3;
    unsigned char bit1 = 1 << (h1 & 7);
    unsigned int w2 = h2 >> 3;
    unsigned char bit2 = 1 << (h2 & 7);
    unsigned int w3 = h3 >> 3;
    unsigned char bit3 = 1 << (h3 & 7);
    unsigned int w4 = h4 >> 3;
    unsigned char bit4 = 1 << (h4 & 7);
//    unsigned int w5 = h5 >> 3;
//    unsigned char bit5 = 1 << (h5 & 7);



    unsigned char t1 = tex1Dfetch(tex_bloom, w1); /* SPEED:  Slowest part */
    if (t1 & bit1) {
        unsigned char t2 = tex1Dfetch(tex_bloom, w2);
        if (t2 & bit2) {
            unsigned char t3 = tex1Dfetch(tex_bloom, w3);
            if (t3 & bit3) {
                unsigned char t4 = tex1Dfetch(tex_bloom, w4);
                if (t4 & bit4) {
//                unsigned char t5 = tex1Dfetch(tex_bloom, w5);
//                if (t5 & bit5) {
                    unsigned int hh5 = (hval + 7 * hval2) & BLOOMMASK;
                    unsigned int h6 = (hval + 3 * hval2) & BLOOMMASK;
                    unsigned int h7 = ((hval << 1) + hval2) & BLOOMMASK;
                    unsigned int h8 = ((hval << 2) + hval2) & BLOOMMASK;
//                    unsigned int h10 = (hval * 11 + hval2) & BLOOMMASK;
                    device_set_bit(hh5, dev_reverse_bloom);
                    device_set_bit(h6, dev_reverse_bloom);
                    device_set_bit(h7, dev_reverse_bloom);
                    device_set_bit(h8, dev_reverse_bloom);
//                    device_set_bit(h10, dev_reverse_bloom);

                    /* If we hit, annotate in a bit vector */
                    /* SPEED:  If we start doing a lot of matches, do this in local
                     * memory and flush all 64 bytes out to main memory.
                     * Not needed yet. */
                    device_set_bit(i, dev_positions_matched);
                }}
//            }
        }
    }
}

__global__ void filterPatterns(unsigned char *d_a,
                               unsigned int *d_b,
                               unsigned int *dev_patterns_matched,
                               unsigned int starting_offset)
{
  /* SPEED:  Copy into local memory coalescing and then do this
   * all locally. */
    int i = starting_offset + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.x;

    int char_offset = i * (HASH_LEN + 1); /* Skip \n */

    unsigned int hval = 0, hval2 = 0;

    for (int j = 0; j < HASH_LEN; j++) {
        hval = rol32(hval, 1);
        hval2 = rol32(hval2, 3);
        unsigned int sbv = sbox[d_a[char_offset + j]];
        hval ^= sbv;
        hval2 ^= sbv;
    }
    unsigned int h5 = (hval + 7 * hval2) & BLOOMMASK;
    unsigned int h6 = (hval + 3 * hval2) & BLOOMMASK;
    unsigned int h7 = ((hval << 1) + hval2) & BLOOMMASK;
    unsigned int h8 = ((hval << 2) + hval2) & BLOOMMASK;
//    unsigned int h10 = (hval * 11 + hval2) & BLOOMMASK;

    if (device_is_bit_set(h5, d_b)) {
        if (device_is_bit_set(h6, d_b)) {
            if (device_is_bit_set(h7, d_b)) {
                if (device_is_bit_set(h8, d_b)) {
//                    if (device_is_bit_set(h10, d_b)) {
                        device_set_bit(i, dev_patterns_matched);
//                    }
                }
            }
        }
    }
}

void checkReportCudaStatus(const char *name) {
    hipError_t err = hipGetLastError();
    printf("CudaStatus %s: ", name);
    if (err) printf("Error: %s\n", hipGetErrorString(err));
    else printf("Success\n");
}

void exitOnError(const char *name, hipError_t err) {
    if (err) {
        if (err) printf("%s Error: %s\n", name, hipGetErrorString(err));
        exit(-1);
    }
}


size_t filetodevice(char *filename, void **devMemPtr)
{
    size_t filesize;
    int f = getfile(filename, &filesize);
    if (f == -1) {
        perror(filename);
        exit(-1);
    }
    char *buf = (char *)mmap(NULL, filesize, PROT_READ,  MAP_FILE | MAP_SHARED, f, 0);
    if (!buf) {
        perror("filetodevice mmap failed");
        exit(-1);
    }
    filesize = min((unsigned long long)filesize, (unsigned long long)FILE_MAX);
    posix_madvise(buf, filesize, POSIX_MADV_SEQUENTIAL);

    printf("filesize = %lu\n", filesize);

    //exitOnError("hipHostMalloc", hipHostMalloc(&pinnedBuf, filesize));

    //memcpy(pinnedBuf, buf, filesize);

    exitOnError("hipMalloc",
		hipMalloc(devMemPtr, filesize + HASH_LEN));

    exitOnError("hipMemcpy",
		hipMemcpy(*devMemPtr, buf, filesize, hipMemcpyHostToDevice));
    munmap(buf, filesize);

    close(f);
    return filesize;
}


void bvDump(char *bloomname, unsigned int *dev_bloom, unsigned int bits) {
    printf("bvDump %s\n", bloomname);
    unsigned int *blooms = (unsigned int *)malloc(bits/8);
    hipMemcpy(blooms, dev_bloom, bits/8, hipMemcpyDeviceToHost);
    for (int i = 0; i < bits; i++) {
        if (is_bit_set(i, blooms)) {
            printf("%d\n", i);
        }
    }
    free(blooms);
}

void printpositions(char *filename,
                    unsigned int *bv,
                    unsigned int file_ints)
{
    size_t filesize;
    int f = getfile(filename, &filesize);
    if (f == -1) {
        perror(filename);
        exit(-1);
    }
    char *buf = pinnedBuf;

    filesize = min((unsigned long long)filesize, (unsigned long long)FILE_MAX);

    int prev_end = -1;
    for (int i = 0; i < file_ints; i++) {
        if (bv[i]) {
	        for (int j = ffs(bv[i]) - 1; j < 32; j++) {
	            int offset = i*32 + j;
	            if (is_bit_set(offset, bv)) {
	                /* Find end of previous line */
	                if (offset > prev_end && buf[offset] != '\n') {
                        char *sol = ((char*)memrchr(buf, '\n', offset));
                        int start_line;
                        if (sol) {
                            start_line = sol - buf;
                        } else {
                            start_line = 0;
                        }

	                    int end_line;
                        char *eol = (char*)memchr(buf + offset, '\n', filesize - offset + 1);
                        end_line = eol - buf;
                        j += end_line - offset;
	                    if (buf[start_line] == '\n') start_line++;
	                    fwrite(buf + start_line, 1, end_line - start_line, stdout);
	                    fputc('\n', stdout);
	                    prev_end = end_line;
	                }
	            }
	        }
        }
    }

    close(f);
}

void printpatterns(char **patterns,
                   int *lengths,
                   unsigned int *bv,
                   unsigned int file_ints,
                   char* out_filename)
{
    FILE *out = fopen(out_filename, "w");
    if (!out) {
        perror("Error opening patterns output file");
        exit(-1);
    }

    for (int i = 0; i < file_ints; i++) {
        if (!bv[i]) {
            continue;
        }
        unsigned qw = bv[i];
        int offset = i << 5;
        for (unsigned mask = 1; mask != 0; mask = mask << 1, offset++) {
            if (qw & mask) {
                fwrite(patterns[offset], 1, lengths[offset], out);
                fputc('\n', out);
            }
        }
    }
    fclose(out);
}

int dimPick(dim3 &dimGrid,
            dim3 &dimBlock,
            int numthreads,
            int blocksize)
{
    unsigned int blocks_y = 1;
    unsigned int blocks_x = 1;
    unsigned int threads_1d = numthreads % blocksize;

    if (numthreads > (256 * blocksize)) {
        blocks_y = numthreads / (256 * blocksize);
        blocks_x = 256;
        threads_1d = blocksize;
    } else if (numthreads > blocksize) {
        blocks_x = numthreads / blocksize;
        threads_1d = blocksize;
    }

    unsigned int threads_used = blocks_y * blocks_x * threads_1d;
    numthreads -= threads_used;
    //printf("dimPick %d %d %d\n", blocks_y, blocks_x, threads_1d);
    dimGrid = dim3(blocks_x, blocks_y);
    dimBlock = dim3(threads_1d);
    return threads_used;
}


void setup_bloom_search(int grepsize,
                        unsigned char *dev_greps,
                        unsigned int *dev_bloom)
{
    exitOnError("setup_bloom_search cudaMemSet dev_bloom = 0",
	            hipMemset(dev_bloom, 0, BLOOMBITS/8));

    int numthreads = grepsize / (HASH_LEN + 1);
    unsigned int char_offset = 0;
    dim3 dimGrid, dimBlock;

    while (numthreads > 0) {
        unsigned int tu = dimPick(dimGrid, dimBlock, numthreads, BLOCK_SIZE);
        printf("Executing grepSetup (%d,%d,%d)\n", dimGrid.x, dimGrid.y, dimBlock.x);
        grepSetup<<<dimGrid, dimBlock>>>(dev_greps, dev_bloom, char_offset);
        checkReportCudaStatus("grepSetup kernel");
        numthreads -= tu;
        char_offset += tu;
    }
}


/*
void executeGrep(int filesize,
                 unsigned char *dev_chars,
                 unsigned int *dev_bloom,
                 unsigned int *dev_reverse_bloom,
                 unsigned int *dev_positions_matched)
{
    int numthreads = filesize;
    unsigned int char_offset = 0;
    dim3 dimGrid, dimBlock;

    exitOnError("executeGrep cudaMemSet dev_reverse_bloom = 0",
	            hipMemset(dev_reverse_bloom, 0, BLOOMBITS/8));


    printf("Executing grep on %d\n", filesize);
    while (numthreads > 0) {
        unsigned int tu = dimPick(dimGrid, dimBlock, numthreads, BLOCK_SIZE);
        printf("Executing GrepKernel (%d,%d,%d) @ %u\n", dimGrid.x, dimGrid.y, dimBlock.x, char_offset);
        GrepKernel<<<dimGrid, dimBlock>>>(dev_chars, dev_bloom, dev_reverse_bloom,
                                          dev_positions_matched, char_offset, filesize);
        numthreads -= tu;
        char_offset += tu;
    }
}
 */


void executeGrepOverlap(char *filename,
                        unsigned char **devMemPtr,
                        unsigned int *dev_bloom,
                        unsigned int *dev_reverse_bloom,
                        unsigned int *dev_positions_matched)
{
    size_t filesize;
    int f = getfile(filename, &filesize);
    if (f == -1) {
        perror(filename);
        exit(-1);
    }
    filesize = min((unsigned long long)filesize, (unsigned long long)FILE_MAX);

    exitOnError("hipHostMalloc", hipHostMalloc(&pinnedBuf, filesize));

    exitOnError("hipMalloc",
		hipMalloc(devMemPtr, filesize + HASH_LEN));
    unsigned char *dev_chars = *devMemPtr;

    int numthreads;
    dim3 dimGrid, dimBlock;

    exitOnError("executeGrep cudaMemSet dev_reverse_bloom = 0",
	            hipMemset(dev_reverse_bloom, 0, BLOOMBITS/8));

    hipStream_t streams[NR_STREAMS];

    for (int i = 0; i < NR_STREAMS; i++) {
        exitOnError("hipStreamCreate",
            hipStreamCreate(&streams[i]));
    }

    int size = filesize / NR_STREAMS;

    int fd = open(filename, O_RDONLY);

    for (int i = 0; i < NR_STREAMS; i++) {
        unsigned offset = i * size;
        if (i == NR_STREAMS - 1) {
            size = filesize - i * size;
        }
        numthreads = size;

        printf("Executing grep on %d\n", size);

        read(fd, pinnedBuf + offset, size);

        exitOnError("hipMemcpyAsync",
            hipMemcpyAsync(dev_chars + offset, pinnedBuf + offset, size, hipMemcpyHostToDevice, streams[i]));

        unsigned int char_offset = 0;
        while (numthreads > 0) {
            unsigned int tu = dimPick(dimGrid, dimBlock, numthreads, BLOCK_SIZE);
            printf("Executing GrepKernel (%d,%d,%d) @ %u\n", dimGrid.x, dimGrid.y, dimBlock.x, (offset + char_offset));
            GrepKernel<<<dimGrid, dimBlock, 0, streams[i]>>>(dev_chars, dev_bloom, dev_reverse_bloom,
                                                             dev_positions_matched, offset + char_offset, size);
            checkReportCudaStatus("GrepKernel");
            numthreads -= tu;
            char_offset += tu;
        }
    }

    close(f);
    close(fd);

    hipDeviceSynchronize();

    for (int i = 0; i < NR_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
}

void executePatternFiltering(int grepsize,
                             unsigned char *dev_greps,
                             unsigned int *dev_reverse_bloom,
                             unsigned int *dev_patterns_matched)
{
    int numthreads = grepsize / (HASH_LEN + 1);
    printf("NUMTHREADS = %d\n", numthreads);
    unsigned int line_offset = 0;
    dim3 dimGrid, dimBlock;

    while (numthreads > 0) {
        unsigned int tu = dimPick(dimGrid, dimBlock, numthreads, BLOCK_SIZE);
        printf("Executing pattern filtering (%d,%d,%d)\n", dimGrid.x, dimGrid.y, dimBlock.x);
        filterPatterns<<<dimGrid, dimBlock>>>(dev_greps, dev_reverse_bloom, dev_patterns_matched, line_offset);
        checkReportCudaStatus("filterPatterns kernel");
        numthreads -= tu;
        line_offset += tu;
    }
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        cerr << "usage: ./greptest truncatedPatternsFile corpus fullPatternsFile outPatternsFile" << endl;
        return -1;
    }

    char *searchfile = argv[1];
    char *infile = argv[2];
    char *full_patterns_file = argv[3];
    char *out_patterns_file = argv[4];

    unsigned char *dev_chars;
    unsigned char *dev_greps;
    /* Bit vectors */
    unsigned int *dev_bloom, *dev_reverse_bloom, *dev_positions_matched, *dev_patterns_matched;

    timing_stamp("start", false);

    size_t grepsize = filetodevice(searchfile, (void **)&dev_greps);
    exitOnError("hipMalloc dev_bloom",
		hipMalloc((void **)&dev_bloom, BLOOMBITS/8));
    exitOnError("hipMalloc dev_reverse_bloom",
		hipMalloc((void **)&dev_reverse_bloom, BLOOMBITS/8));

    setup_bloom_search(grepsize, dev_greps, dev_bloom);

    /* Bind the bloom filter to a texture */
    exitOnError("hipBindTexture tex_bloom to dev_bloom",
		hipBindTexture(NULL, tex_bloom, dev_bloom, BLOOMBITS/8));

    //bvDump(searchfile, dev_bloom, BLOOMBITS);

    //index patterns by line
    int nr_patterns = grepsize / (HASH_LEN + 1);
    char **patterns = new char*[nr_patterns];
    int *lengths = new int[nr_patterns];
    ifstream in_patterns(full_patterns_file);
    printf("nr_patterns = %d\n", nr_patterns);
    for (int i = 0; i < nr_patterns; i++) {
        char line[1001];
        in_patterns.getline(line, 1000);
        patterns[i] = new char[strlen(line) + 1];
        memcpy(patterns[i], line, strlen(line) + 1);
        lengths[i] = strlen(patterns[i]);
    }
    in_patterns.close();

    timing_stamp("setup complete", false);

    printf("GPUGrep opening %s\n", infile);
    size_t filesize;
    int f = getfile(infile, &filesize);
    if (f == -1) {
        perror(infile);
        exit(-1);
    }

    exitOnError("hipMalloc dev_positions_matched",
		hipMalloc((void **)&dev_positions_matched, filesize / 8 + 1));
    exitOnError("hipMemset dev_positions_matched = 0",
		hipMemset(dev_positions_matched, 0, filesize/8 + 1));

    timing_stamp("posmatch init", false);

    printf("\nPhase 3:  Executing kernel\n");
    executeGrepOverlap(infile, &dev_chars, dev_bloom, dev_reverse_bloom, dev_positions_matched);

    hipDeviceSynchronize();
    timing_stamp("grep done", false);
    checkReportCudaStatus("Grep Kernel");

    exitOnError("hipMalloc dev_patterns_matched",
		hipMalloc((void **)&dev_patterns_matched, nr_patterns / 8));
    exitOnError("hipMemset dev_patterns_matched = 0",
		hipMemset(dev_patterns_matched, 0, nr_patterns / 8));


    executePatternFiltering(grepsize, dev_greps, dev_reverse_bloom, dev_patterns_matched);
    hipDeviceSynchronize();
    hipFree(dev_greps);

    timing_stamp("patterns filtering done", false);

    /* Idea:
     * Record array of bit positions to check + chars;
     * Sort that array.
     * Divvy up the array to threads.  Compute min, max of the bit vector
     * address space accessed by that array, and pull that part of the BV array
     * (as much as fits?) into local shared memory.  Check in parallel, and
     * issue atomic increments to the set bit positions into a global count
     * array (presumably somewhat rare???).
     * If that takes too long, then output the maps of counts and
     * char offsets, sort that, merge, and then do the bit sets. */
    /* But :  radixSort only gets 20 MElements/sec;  very possibly
     * slower than what we're already doing. */

#if 1
    printf("\nPhase 4:  Copying results to host memory.\n");

    unsigned int *host_positions_matched = (unsigned int *)malloc(filesize / 8);
    unsigned int *host_patterns_matched = (unsigned int *)malloc(grepsize);

    exitOnError("hipMemcpy corpus results to host",
		hipMemcpy(host_positions_matched, dev_positions_matched,
			   filesize / 8, hipMemcpyDeviceToHost));
    exitOnError("hipMemcpy pattern results to host",
		hipMemcpy(host_patterns_matched, dev_patterns_matched,
			   grepsize / (HASH_LEN + 1) / 8, hipMemcpyDeviceToHost));

    timing_stamp("copyout done", false);
#if 1
    printpositions(infile, host_positions_matched, filesize / 32);
    printpatterns(patterns, lengths, host_patterns_matched, grepsize / (HASH_LEN + 1) / 32, out_patterns_file);
    timing_stamp("printout done", false);
#endif
    printf("\n");
    free(host_positions_matched);
    free(host_patterns_matched);
#endif
    hipFree(dev_bloom);
    hipFree(dev_reverse_bloom);
    hipFree(dev_chars);
    hipFree(dev_positions_matched);
    timing_stamp("cleanup done", true);
    timing_report();

    struct hipDeviceProp_t cdp;
    hipGetDeviceProperties(&cdp, 0);
    printf("\ndeviceOverlap = %d\n", cdp.deviceOverlap);
}

